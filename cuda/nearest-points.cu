#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 4
const int lowerLimit = -100;
const int upperLimit = 100;

struct Lock {

    int *d_state;

    // --- Constructor
    Lock(void) {
        int h_state = 0;                                        // --- Host side lock state initializer
        hipMalloc((void **)&d_state, sizeof(int));  // --- Allocate device side lock state
        hipMemcpy(d_state, &h_state, sizeof(int), hipMemcpyHostToDevice); // --- Initialize device side lock state
    }

    // --- Destructor
    __host__ __device__ ~Lock(void) { 
    #if !defined(__HIPCC__)
        hipFree(d_state); 
    #else

    #endif  
    }

    // --- Lock function
    __device__ void lock(void) { while (atomicCAS(d_state, 0, 1) != 0); }

    // --- Unlock function
    __device__ void unlock(void) { atomicExch(d_state, 0); }
};


struct point
{
    double x;
    double y;
    
    point(double x, double y) : x(x), y(y) {}
};

__global__ void findNearestPointWithLocks(point* points, point* nearestPoints, double* nearestPointsDistances, int* blockLocks)
{
    if (blockIdx.x == 0)
    {
        int id = blockIdx.x;
        printf("Lock at %d with value %d\n", threadIdx.x, blockLocks[id]);
        while (atomicCAS(&blockLocks[id], 0, 1) != 0) ; // lock();
        printf("Thread %d aquired the lock\n", threadIdx.x);
        nearestPointsDistances[id] += 1;
        atomicExch(&blockLocks[id], 0); // unlock();
    }
}

__global__ void findNearestPoint(point* points, point* nearestPoints, double* nearestPointsDistances)
{
    int blockId = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
    int threadId = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x + blockId * (blockDim.x * blockDim.y * blockDim.z);    
    
    point A = points[threadId];
    for (int i = 0; i < N; i++) 
    {
        if (threadId == i) continue;
        point B = points[i];
        double distance = sqrt((A.x - B.x) * (A.x - B.x) + (A.y - B.y) * (A.y - B.y));
        if (distance < nearestPointsDistances[threadId]) 
        {
            //printf("Closest to point (%f,%f) is (%f,%f) at euclidian distance of %f\n", A.x, A.y, B.x, B.y, distance);
            nearestPointsDistances[threadId] = distance;
            nearestPoints[threadId] = B;
        }
    }
}

int main () {
    srand(time(NULL));
    hipDeviceReset();

    dim3 gridSize(1);
    dim3 blockSize(N);

    // Declaring and initializing host variables
    int* blockLocks = (int*) malloc(N * sizeof(int));
    point* points = (point*) malloc(N * sizeof(point));
    point* nearestPoints = (point*) malloc(N * sizeof(point));
    double* nearestPointsDistances = (double*) malloc(N * sizeof(double));
    for (int i = 0; i < N; i++)
    {
        blockLocks[i] = 0;
        points[i].x = lowerLimit + rand() % ((upperLimit - lowerLimit) + 1);
        points[i].y = lowerLimit + rand() % ((upperLimit - lowerLimit) + 1);
        nearestPointsDistances[i] = (upperLimit - lowerLimit) + 10; // The maximum distance plus an extra 10
        //printf("(%f,%f)\n", points[i].x, points[i].y);
    }

    // Create device variables and allocate memory on device
    point* points_d;
    point* nearestPoints_d;
    int* blockLocks_d;
    double* nearestPointsDistances_d;
    hipMalloc(&points_d, N * sizeof(point));
    hipMalloc(&nearestPoints_d, N * sizeof(point));
    hipMalloc(&blockLocks_d, N * sizeof(int));
    hipMalloc(&nearestPointsDistances_d, N * sizeof(double));

    // Copy values to devices
    hipMemcpy(points_d, points, N * sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(nearestPointsDistances_d, nearestPointsDistances, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(blockLocks_d, blockLocks, N * sizeof(int), hipMemcpyHostToDevice);

    // Execute kernel
    findNearestPoint<<<gridSize, blockSize>>>(points_d, nearestPoints_d, nearestPointsDistances_d);
    
    // Wait till every thread has finished
    hipDeviceSynchronize();

    // Retrieve values from device memory
    hipMemcpy(nearestPointsDistances, nearestPointsDistances_d, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(nearestPoints, nearestPoints_d, N * sizeof(point), hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; i++)
    {
        printf("Closest to point (%f,%f) is (%f,%f) at euclidian distance of %f\n",
        points[i].x, points[i].y, nearestPoints[i].x, nearestPoints[i].y, nearestPointsDistances[i]);
    }

    // Free all resources
    hipDeviceReset();

    return 0;
}