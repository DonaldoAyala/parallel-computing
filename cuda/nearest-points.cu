#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 30
const int lowerLimit = -200;
const int upperLimit = 200;

struct Lock {

    int *d_state;

    // --- Constructor
    Lock(void) {
        int h_state = 0;                                        // --- Host side lock state initializer
        hipMalloc((void **)&d_state, sizeof(int));  // --- Allocate device side lock state
        hipMemcpy(d_state, &h_state, sizeof(int), hipMemcpyHostToDevice); // --- Initialize device side lock state
    }

    // --- Destructor
    __host__ __device__ ~Lock(void) {
    #if !defined(__HIPCC__)
        hipFree(d_state); 
    #else

    #endif  
    }

    // --- Lock function
    __device__ void lock(void) { while (atomicCAS(d_state, 0, 1) != 0); }

    // --- Unlock function
    __device__ void unlock(void) { atomicExch(d_state, 0); }
};


struct point
{
    double x;
    double y;
    
    point(double x, double y) : x(x), y(y) {}
};

__global__ void findNearestPointWithLocks(point* points, point* nearestPoints, double* nearestPointsDistances, int* blockLocks)
{
    if (blockIdx.x == 0)
    {
        int id = blockIdx.x;
        while (atomicCAS(&blockLocks[id], 0, 1) != 0) ; // lock();
        nearestPointsDistances[id] += 1;
        atomicExch(&blockLocks[id], 0); // unlock();
    }
}

__global__ void findNearestPoint(point* points, point* nearestPoints, double* nearestPointsDistances)
{
    int blockId = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
    int threadId = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x + blockId * (blockDim.x * blockDim.y * blockDim.z);    
    
    point A = points[threadId];
    for (int i = 0; i < N; i++) 
    {
        if (threadId == i) continue;
        point B = points[i];
        double distance = sqrt((A.x - B.x) * (A.x - B.x) + (A.y - B.y) * (A.y - B.y));
        if (distance < nearestPointsDistances[threadId]) 
        {
            nearestPointsDistances[threadId] = distance;
            nearestPoints[threadId] = B;
        }
    }
}

int main () {
    srand(time(NULL));
    hipDeviceReset();

    dim3 gridSize(1);
    dim3 blockSize(N);

    // Declaring and initializing host variables
    int* blockLocks = (int*) malloc(N * sizeof(int));
    point* points = (point*) malloc(N * sizeof(point));
    point* nearestPoints = (point*) malloc(N * sizeof(point));
    double* nearestPointsDistances = (double*) malloc(N * sizeof(double));
    for (int i = 0; i < N; i++)
    {
        blockLocks[i] = 0;
        points[i].x = lowerLimit + rand() % ((upperLimit - lowerLimit) + 1);
        points[i].y = lowerLimit + rand() % ((upperLimit - lowerLimit) + 1);
        nearestPointsDistances[i] = (upperLimit - lowerLimit) + 10; // The maximum distance plus an extra 10
    }

    // Create device variables and allocate memory on device
    point* points_d;
    point* nearestPoints_d;
    int* blockLocks_d;
    double* nearestPointsDistances_d;
    hipMalloc(&points_d, N * sizeof(point));
    hipMalloc(&nearestPoints_d, N * sizeof(point));
    hipMalloc(&blockLocks_d, N * sizeof(int));
    hipMalloc(&nearestPointsDistances_d, N * sizeof(double));

    // Copy values to devices
    hipMemcpy(points_d, points, N * sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(nearestPointsDistances_d, nearestPointsDistances, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(blockLocks_d, blockLocks, N * sizeof(int), hipMemcpyHostToDevice);

    // Execute kernel
    findNearestPoint<<<gridSize, blockSize>>>(points_d, nearestPoints_d, nearestPointsDistances_d);
    
    // Wait till every thread has finished
    hipDeviceSynchronize();

    // Retrieve values from device memory
    hipMemcpy(nearestPointsDistances, nearestPointsDistances_d, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(nearestPoints, nearestPoints_d, N * sizeof(point), hipMemcpyDeviceToHost);

    // Search for the nearest points
    point pointA = points[0];
    point pointB = nearestPoints[0];
    double nearestPointsDistance = nearestPointsDistances[0];
    for (int i = 1; i < N; i++)
    {
        if (nearestPointsDistances[i] < nearestPointsDistance) 
        {
            nearestPointsDistance = nearestPointsDistances[i];
            pointA = points[i];
            pointB = nearestPoints[i];
        }
    }

    // Print number of points
    printf("%d\n", N);
    // Print points coordinates
    for (int i = 0; i < N; i++) 
    {
        printf("%f,%f\n", points[i].x, points[i].y);
    }
    // Print nearest poins and distance
    printf("%f,%f\n", pointA.x, pointA.y);
    printf("%f,%f\n", pointB.x, pointB.y);
    printf("%f\n", nearestPointsDistance);

    // Free all resources
    hipDeviceReset();

    return 0;
}